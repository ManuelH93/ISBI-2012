#include "hip/hip_runtime.h"
#include <cstdint>

// sets of fg/bg pixels in python not sorted, so slightly different result
// here, total sum over array should be identical
__device__ void _normConsensusArray(
    unsigned idx, unsigned idy, unsigned idz,
    const float inPred[][DATAZSIZE][DATAYSIZE][DATAXSIZE],
    float outCons[][NSY][NSX][DATAZSIZE][DATAYSIZE][DATAXSIZE],
    float outConsCnt[][NSY][NSX][DATAZSIZE][DATAYSIZE][DATAXSIZE])
{
    if ((idx < (DATAXSIZE)) &&
        (idy < (DATAYSIZE)) &&
        (idz < (DATAZSIZE))){

        unsigned int mid = int((PSX*PSY*PSZ)/2);
        if(inPred[mid][idz][idy][idx] <= TH)
            return;

    for(int z = 0; z < NSZ; z++) {
        for(int y = 0; y < NSY; y++) {
            for(int x = 0; x < NSX; x++) {
                if(outConsCnt[z][y][x][idz][idy][idx] != 0)
                    outCons[z][y][x][idz][idy][idx] = outCons[z][y][x][idz][idy][idx]/outConsCnt[z][y][x][idz][idy][idx];
            }
        }
    }
    }
}


// device function to set the 3D volume
__global__ void normConsensusArray(
    const float inPred[][DATAZSIZE][DATAYSIZE][DATAXSIZE],
    float outCons[][NSY][NSX][DATAZSIZE][DATAYSIZE][DATAXSIZE],
    float outConsCnt[][NSY][NSX][DATAZSIZE][DATAYSIZE][DATAXSIZE])
{
    // pixel for this thread: idz, idy, idx
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;

    _normConsensusArray(idx, idy, idz, inPred, outCons, outConsCnt);
}
